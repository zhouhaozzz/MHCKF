#pragma once
#include <hip/hip_runtime.h>


#ifdef CUDA
#include "function_cuda.cuh"

__global__ void LSQ_CUDA::transpose_cuda(double* T, double* A_T, int raw, int col) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < raw && j < col) {
		T[j * raw + i] = A_T[i * col + j];
	}
}

__global__ void LSQ_CUDA::multiply_cuda(double* A, double* B, double* C, int rowA, int colA, int rowB, int colB)
{
    __shared__ double sA[TILE_WIDTH][TILE_WIDTH];
    __shared__ double sB[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;  int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;
    double Cvalue = 0.0;

    for (int t = 0; t < (colA - 1) / TILE_WIDTH + 1; ++t) {
        if (Row < rowA && t * TILE_WIDTH + tx < colA)
            sA[ty][tx] = A[Row * colA + t * TILE_WIDTH + tx];
        else
            sA[ty][tx] = 0.0;
        if (Col < colB && t * TILE_WIDTH + ty < rowB)
            sB[ty][tx] = B[(t * TILE_WIDTH + ty) * colB + Col];
        else
            sB[ty][tx] = 0.0;
        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; ++i)
            Cvalue += sA[ty][i] * sB[i][tx];
        __syncthreads();
    }

    if (Row < rowA && Col < colB)
        C[Row * colB + Col] = Cvalue;
}

#endif // CUDA