#include "LSQ_cuda.cuh"

#ifdef CUDA
std::vector<double> LSQ_CUDA::normal_equation(const std::vector<std::vector<double>>& M, const std::vector<double>& H)
{
    // ��ȡ��ǰʱ����Ϊ��ʼʱ���
    auto start = std::chrono::high_resolution_clock::now(); 
    
    int m = M.size();
    int n = M[0].size();    
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((n + 15) / 16, (m + 15) / 16);
    double* d_M;
    double* d_M_T;
    double* d_H;
    double* A;
    hipMalloc(&d_M, n * m * sizeof(double));
    hipMalloc(&d_H, m * sizeof(double));
    hipMalloc(&d_M_T, m * n * sizeof(double));
    hipMalloc(&A, m * n * sizeof(double));

    hipMemcpy(d_M, M.data(), n * m * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_H, H.data(), m * sizeof(double), hipMemcpyHostToDevice);

    LSQ_CUDA::transpose_cuda << <numBlocks, threadsPerBlock >> > (d_M, d_M_T, m, n);
    LSQ_CUDA::multiply_cuda << <numBlocks, threadsPerBlock >> > (d_M_T, d_M, A, m, n, n, m);

    //LSQ_CUDA::inverse_cuda << <numBlocks, threadsPerBlock >> > (d_M, d_M_T, m, n);


    hipFree(d_M);
    hipFree(d_H);

    // ��ȡ��ǰʱ����Ϊ����ʱ���
    auto end = std::chrono::high_resolution_clock::now();
    // ���㺯��ִ��ʱ��
    std::chrono::duration<double> duration = end - start;
    std::cout << "cuda execution time: " << duration.count() << " seconds" << std::endl;

    return H;
}
#endif // CUDA
